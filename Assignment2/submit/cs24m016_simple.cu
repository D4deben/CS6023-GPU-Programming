#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void dkernel(long int *matrix, long int *filter, long int *result, int h, int w, int c, int r, int s, int k)
 {
       // sample kernel you can use your own kernel
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
//     printf("hey welcome to my kernel, %d %d\n",row,col);
    if(row<h*k && col<w){
    int f= row/h;
    int mat_row=row%h;
    int sum=0;
    
    for (int ch = 0; ch < c; ch++) {
            for (int fi = 0; fi < r; fi++) {
                for (int fj = 0; fj < s; fj++) {
                    int a = mat_row + fi - (r / 2);
                    int b = col + fj - (s / 2);

                    if (a >= 0 && a < h && b >= 0 && b < w) {
                        int mat_index = (ch * h + a) * w + b;
                        int filter_index = ((f * c + ch) * r + fi) * s + fj;
                        sum += matrix[mat_index] * filter[filter_index];
                    }
                }
            }
        }
        result[row * w + col] = sum;
        //if(f==1) printf("%d\n", sum);
    }
}

int main(int argc, char **argv)
{
    int h, w, c;
    
    cin >> h >> w >> c;
    long int *h_mat = new long int[h * w * c];
    for (long int i = 0; i < h * w * c; i++)
    {
        cin >> h_mat[i];
    }

    int cf, r, s, k;
    cin >> cf >> r >> s >> k;

    long int *h_filter = new long int[r * s * c * k];
    for (long int i = 0; i < r * s * c * k; i++)
    {
        cin >> h_filter[i];
    }
    long int *h_ans = new long int[h * w * k];

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     **/

    auto start = std::chrono::high_resolution_clock::now(); // keep it just before the kernel launch

    /****************************************************Start Here***********************************************************/

    /**
        Do device allocations, kernel launches and copying everything here
        and the final answer should be stored back in h_ans, use cudaFree to free up the allocated memory on GPU
    */
    
    long int *d_mat, *d_filter, *d_ans;
    int BLOCK_SIZE= 32;
    hipMalloc(&d_ans, h * w * k * sizeof(long int));
    hipMalloc(&d_filter, r * s * c * k * sizeof(long int));
    hipMalloc(&d_mat, h * w * c * sizeof(long int));
   // cout<<"mem allocated"<<endl;
    hipMemcpy(d_mat, h_mat, h * w * c * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, r * s * c * k * sizeof(long int), hipMemcpyHostToDevice);
   // cout<<"coped to gpu"<<endl;

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((w + BLOCK_SIZE-1) / BLOCK_SIZE, ((h*k) + BLOCK_SIZE-1) / BLOCK_SIZE);
//	cout<<"calling kernel"<<endl;
    dkernel<<<gridDim, blockDim>>>(d_mat, d_filter, d_ans, h, w, c, r, s, k);
//	cout<<"kernel call done"<<endl;
    hipMemcpy(h_ans, d_ans, h * w * k * sizeof(long int), hipMemcpyDeviceToHost);
  //  cout<<"copied to cpu"<<endl;
    hipFree(d_mat);
    hipFree(d_filter);
    hipFree(d_ans);
	
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    auto end = std::chrono::high_resolution_clock::now(); // keep it just after the kernel launch
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    hipDeviceSynchronize();
    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < h * k; i++)
        {
            for (long int j = 0; j < w; j++)
            {
                file << h_ans[i * w + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}
