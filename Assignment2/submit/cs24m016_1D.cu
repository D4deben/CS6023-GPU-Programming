#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>


using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void dkernel(long int *matrix, long int *filter, long int *result, int h, int w, int c, int r, int s, int k)
{
    // Calculate unique thread index in 1D grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Load filter into shared memory in chunks (reduces shared memory usage)
    extern __shared__ long int mini_filter[];

    for (int i = threadIdx.x; i < c * r * s * k; i += blockDim.x)
    {
        mini_filter[i] = filter[i];
    }

    __syncthreads();

    // Each thread handles one output pixel
    while (tid < h * w * k)
    {
        int col = tid % w;
        int row = (tid / w) % h;
        int f = tid / (h * w); // Filter index

        ll sum = 0; // Use long long to prevent overflow

        for (int ch = 0; ch < c; ch++)
        {
            for (int fi = 0; fi < r; fi++)
            {
                for (int fj = 0; fj < s; fj++)
                {
                    int a = row + fi - (r / 2);
                    int b = col + fj - (s / 2);

                    if (a >= 0 && a < h && b >= 0 && b < w)
                    {
                        int mat_index = (ch * h + a) * w + b;
                        int filter_index = ((f * c + ch) * r + fi) * s + fj;
                        sum += (ll)matrix[mat_index] * (ll)mini_filter[filter_index]; // Prevent overflow
                    }
                }
            }
        }
        result[tid] = (long int)sum;

        tid += total_threads; // Ensure all elements are covered
    }
}

int main(int argc, char **argv)
{
    int h, w, c;

    cin >> h >> w >> c;
    long int *h_mat = new long int[h * w * c];
    for (long int i = 0; i < h * w * c; i++)
    {
        cin >> h_mat[i];
    }

    int cf, r, s, k;
    cin >> cf >> r >> s >> k;

    long int *h_filter = new long int[r * s * c * k];
    for (long int i = 0; i < r * s * c * k; i++)
    {
        cin >> h_filter[i];
    }
    long int *h_ans = new long int[h * w * k];

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     **/

    auto start = std::chrono::high_resolution_clock::now(); // keep it just before the kernel launch

    /****************************************************Start Here***********************************************************/

    long int *d_mat, *d_filter, *d_ans;
    int BLOCK_SIZE = 256; // Optimal for memory coalescing
    int TOTAL_THREADS = h * w * k;
    int GRID_SIZE = (TOTAL_THREADS + BLOCK_SIZE - 1) / BLOCK_SIZE; // Ensuring full coverage

    hipMalloc(&d_ans, h * w * k * sizeof(long int));
    hipMalloc(&d_filter, r * s * c * k * sizeof(long int));
    hipMalloc(&d_mat, h * w * c * sizeof(long int));

    hipMemcpy(d_mat, h_mat, h * w * c * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, r * s * c * k * sizeof(long int), hipMemcpyHostToDevice);

    int sharedMemSize = c * r * s * k * sizeof(long int);

    dkernel<<<GRID_SIZE, BLOCK_SIZE, sharedMemSize>>>(d_mat, d_filter, d_ans, h, w, c, r, s, k);

    hipMemcpy(h_ans, d_ans, h * w * k * sizeof(long int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_filter);
    hipFree(d_ans);

    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    auto end = std::chrono::high_resolution_clock::now(); // keep it just after the kernel launch
    std::chrono::duration<double> elapsed1 = end - start;

    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    hipDeviceSynchronize();
    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < h * k; i++)
        {
            for (long int j = 0; j < w; j++)
            {
                file << h_ans[i * w + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}

